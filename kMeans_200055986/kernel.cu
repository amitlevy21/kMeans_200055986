#include "hip/hip_runtime.h"

#include "kernel.h"

__global__ void movePoints(double *devPoints, // points that were copied to device
	double *devSpeeds,			//speeds that were copied to device
	int numOfPoints,
	int numDims,
	int numThreadsInBlock,		//each thread takes care of one coord of one point
	double dt)					//the differencial for the change of coord
{
	int blockID = blockIdx.x;
	int i;

	//because we optimized num of blocks before calling the calling
	if ((blockID == gridDim.x - 1) && (numOfPoints % blockDim.x <= threadIdx.x)) { return; } //dismiss spare threads

	for (i = 0; i < numDims; i++)
	{
		devPoints[(blockID * numThreadsInBlock + threadIdx.x) * numDims + i] += devSpeeds[(blockID * numThreadsInBlock + threadIdx.x) * numDims + i] * dt;		
	}
}

__global__ void computeDistancesArray(double *devPoints,
	double *devClusters,
	int    numPoints,
	int    numClusters,
	int    numThreadsInBlock,
	int    numDims,
	double *devDistsPointsToClusters)
{
	int i;
	int blockID = blockIdx.x;
	double result = 0;

	if ((blockID == gridDim.x - 1) && (numPoints % blockDim.x <= threadIdx.x)) { return; } //dismiss spare threads

	//each thread computes a distance in a matrix of distances - the distance from each cluster center
	//no need to compute square root of result: (dist1 < dist2) <=> (sqrt(dist1) < sqrt(dist2))
	for (i = 0; i < numDims; i++)
	{
		result += (devPoints[(blockID * numThreadsInBlock + threadIdx.x) * numDims + i] - devClusters[threadIdx.y * numDims + i])
			* (devPoints[(blockID * numThreadsInBlock + threadIdx.x) * numDims + i] - devClusters[threadIdx.y * numDims + i]);
	}

	//this array contains for each point its distance from each cluster
	devDistsPointsToClusters[numPoints * threadIdx.y + (blockID * numThreadsInBlock + threadIdx.x)] = result;
}

__global__ void findMinDistanceForEachPointFromCluster(int numPoints,
	int    numClusters,
	int    numThreadsInBlock,
	double *devDistsPointsToClusters,
	int   *devPToCRelevance)
{
	int i;
	int blockId = blockIdx.x;
	double minIndex = 0;
	double minDistance, tempDistance;

	if ((blockIdx.x == gridDim.x - 1) && (numPoints % blockDim.x <= threadIdx.x)) { return; }  //dismiss spare threads

	minDistance = devDistsPointsToClusters[(numThreadsInBlock * blockId) + threadIdx.x];

	for (i = 1; i < numClusters; i++)
	{
		tempDistance = devDistsPointsToClusters[(numThreadsInBlock * blockId) + threadIdx.x + (i * numPoints)];
		if (minDistance > tempDistance)
		{
			minIndex = i;
			minDistance = tempDistance;
		}
	}

	devPToCRelevance[numThreadsInBlock * blockId + threadIdx.x] = minIndex;
}

hipError_t movePointsWithCuda(double **points,	//cpu points that will be updated with new coords
	double *devPoints, 		//points that were copied to device
	double *devSpeeds,		//speeds that were copied to device
	int numOfPoints,
	int numDims,
	double dt)
{
	hipError_t cudaStatus;
	hipDeviceProp_t devProp; //used to retrieve specs from GPU

	int numBlocks, numThreadsInBlock;

	hipGetDeviceProperties(&devProp, 0); // 0 is for device 0

	//if numOfPoints is bigger the maxThreadsPerBlock this is a problem
	//numThreadsInBlock = devProp.maxThreadsPerBlock / numOfPoints;
	numThreadsInBlock = devProp.maxThreadsPerBlock;
	numBlocks = numOfPoints / numThreadsInBlock;
	
	if (numOfPoints % numThreadsInBlock > 0) { numBlocks++; }

	movePoints<<<numBlocks, numThreadsInBlock>>>(devPoints, devSpeeds, numOfPoints, numDims, numThreadsInBlock, dt);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	/* hipDeviceSynchronize waits for the kernel to finish, and returns
	any errors encountered during the launch*/
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	
	//update the points from gpu to cpu
	cudaStatus = hipMemcpy((void**)points[0], devPoints, numOfPoints * numDims * sizeof(double), hipMemcpyDeviceToHost);

Error:
	return cudaStatus;
}

hipError_t classifyPointsToClusters(double *devPoints,
	double **clusters,
	int     numPoints,
	int     numClusters,
	int		numDims,
	int    *pToCRelevance)
{
	hipError_t cudaStatus;
	hipDeviceProp_t devProp; //used to retrieve specs from GPU

	int numBlocks, numThreadsInBlock;

	hipGetDeviceProperties(&devProp, 0); // 0 is for device 0

	//configuring kernel params
	numThreadsInBlock = devProp.maxThreadsPerBlock / numClusters;
	dim3 dim(numThreadsInBlock, numClusters);
	numBlocks = numPoints / numThreadsInBlock;

	if (numPoints % numThreadsInBlock > 0) { numBlocks++; }

	double *devClusters;
	double *devDistsPointsToClusters = 0;
	int   *devPToCRelevance = 0;

	cudaStatus = hipMalloc((void**)&devClusters, numClusters * numDims * sizeof(double));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devDistsPointsToClusters, numClusters * numPoints * sizeof(double));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devPToCRelevance, numPoints * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input from host memory to GPU buffers.
	cudaStatus = hipMemcpy(devClusters, clusters[0], numClusters * numDims * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//launch kernels//
	computeDistancesArray <<<numBlocks, dim >>> (devPoints, devClusters, numPoints, numClusters, numThreadsInBlock, numDims, devDistsPointsToClusters);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	/* hipDeviceSynchronize waits for the kernel to finish, and returns
	any errors encountered during the launch*/
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	//reconfiguring params for next kernel
	numThreadsInBlock = devProp.maxThreadsPerBlock;
	numBlocks = numPoints / numThreadsInBlock;
	if (numPoints % numThreadsInBlock > 0) { numBlocks++; }

	findMinDistanceForEachPointFromCluster <<<numBlocks, numThreadsInBlock >>> (numPoints, numClusters, numThreadsInBlock, devDistsPointsToClusters, devPToCRelevance);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	/* hipDeviceSynchronize waits for the kernel to finish, and returns
	any errors encountered during the launch*/
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output from GPU buffer to host memory.
	cudaStatus = hipMemcpy(pToCRelevance, devPToCRelevance, numPoints * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


Error:
	hipFree(devClusters);
	hipFree(devDistsPointsToClusters);
	hipFree(devPToCRelevance);

	return cudaStatus;
}
